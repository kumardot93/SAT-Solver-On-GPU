
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <stdbool.h>
#include <time.h>

#define K 3 // K is from K-SAT, currently we are working on 3-SAT
#define THREAD_PER_BLOCK_log2 10

// current Var Limit is 32;


void preProcessing(){

    // removes comment      
    while(getchar() == 'c'){
        while(getchar()!='\n');
    }
     
    getchar();
    char format[100];
    scanf("%s", format);
    if(strcmp(format, "cnf") != 0){    // format assertion
        printf("Format Error, expected cnf but %s was provided\n", format);
        exit(1);
    }    
    printf("Preprocessing Successfull\n");

}

int cpuSolve(int varCount, int clauseCount, int* clauseStore){
    int limit = pow(2, varCount);
    int satCount = 0;    

    for(int perIndex=0; perIndex<limit; perIndex++){
        bool result = true;
        
        for(int i=0; i<clauseCount; i++){
            bool clauseResult = false;
            for(int j=0; j<K; j++){
                int var = clauseStore[K*i + j];
                int absVar = abs(var);
                bool varValue;
                if(var < 0)
                    varValue = !((perIndex >> (absVar-1))&1);
                else 
                    varValue = (perIndex >> (absVar-1))&1;
                clauseResult = clauseResult || varValue;
            }
            result = result  && clauseResult;
        }
        if(result)
            satCount++;

//        if(perIndex%10000 == 0)
//            printf("completed  = %d\n", perIndex);
    } 
    
    return satCount;
}

__global__ void gpuSolver(int varCount, int clauseCount, int limit, int* clauseStore, int *gpu_sat_count){
    bool result = true;
    int perIndex = (blockIdx.x << THREAD_PER_BLOCK_log2) + threadIdx.x;
    
    if(perIndex >= limit)
        return;

    for(int i=0; i<clauseCount; i++){
    bool clauseResult = false;
        for(int j=0; j<K; j++){
            int var = clauseStore[K*i + j];
            int absVar = abs(var);
            bool varValue;
            if(var < 0)
                varValue = !((perIndex >> (absVar-1))&1);
            else
                varValue = (perIndex >> (absVar-1))&1;
            clauseResult = clauseResult || varValue;
        }
        result = result  && clauseResult;
    }
    if(result)
        atomicAdd(gpu_sat_count, 1);
}

int main(int argc, char* argv[]){
    if(argc<2){
        printf("Invalid Options: One options is required to indetity type of execution\n");
        return 1;
    }

    preProcessing();

    int varCount, clauseCount;
    scanf("%d%d", &varCount, &clauseCount);
   
    printf("\nNo. of Variables = %d | No. of clauses = %d\n", varCount, clauseCount); 

	
    // clauses Input
    int *clauseStore = (int*)malloc(sizeof(int)*clauseCount*K);
    
    for(int i=0; i<clauseCount; i++){

        for(int j=0; j<K; j++){ // one clause with K variables
            scanf("%d", clauseStore + (K * i) + j);
        }

        int tmp;
        scanf("%d\n", &tmp);
    }

    clock_t start, end;

    /* for(int i=0; i<clauseCount; i++){
        for(int j=0; j<K; j++){
            printf("%d ", clauseStore[K*i + j]);
        }
        printf("\n");
    } */

    if(strcmp(argv[1], "cpu")==0){  // cpu implementations
        start = clock();
        int satCount =  cpuSolve(varCount, clauseCount, clauseStore);
        end = clock();
        printf("\n\nSAT Count = %d\n", satCount);
    }
    else if(strcmp(argv[1], "gpu") ==0){        // gpu implementations
        int *gpuClauseStore;
        hipMalloc(&gpuClauseStore, sizeof(int)*clauseCount*K);
        hipMemcpy(gpuClauseStore, clauseStore, sizeof(int)*clauseCount*K, hipMemcpyHostToDevice);    
    
        int *gpu_sat_count;
        hipMalloc(&gpu_sat_count, sizeof(int));
        hipMemset(gpu_sat_count, 0, sizeof(int));
        hipDeviceSynchronize();
        int limit = pow(2, varCount);
        int threadPerBlock = pow(2, THREAD_PER_BLOCK_log2);
        int noOfBlock = ceil((float)limit / threadPerBlock);
        
        start = clock();
        gpuSolver<<<noOfBlock, threadPerBlock>>>(varCount, clauseCount, limit, gpuClauseStore, gpu_sat_count);
        hipDeviceSynchronize();
        end = clock();

        int *satCount= (int*)malloc(sizeof(int));
        hipMemcpy(satCount, gpu_sat_count,  sizeof(int), hipMemcpyDeviceToHost);
        printf("\n\nSAT Count = %d\n", *satCount);
    }
    else{
        printf("Invalid Option");
        return 0;
    }

    double executionTime = (double)(end-start)/CLOCKS_PER_SEC;
    printf("execution Time = %lf\n", executionTime);

    return 0;
}
